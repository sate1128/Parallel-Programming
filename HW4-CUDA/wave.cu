/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

__global__ void calc(float* finalval,int tpoints, int nsteps)
{
	int i;
	float x, values, newval, oldval;
	int tx = blockIdx.x * blockDim.x + threadIdx.x;

	/* Calculate initial values based on sine curve */
	x = float(tx - 1) / (tpoints - 1);
	values = sin (2.0 * PI * x);
	
	/* Initialize old values array */
	oldval = values;
	
	if ((tx == 1) || (tx  == tpoints))
		values = 0.0;
	else{
		for (i = 1; i<= nsteps; i++) {
			newval = 1.82 * values - oldval;
			/* Update old values with new values */
			oldval = values;
			values = newval;
		}
	}
	finalval[tx] = values;
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal(float* values)
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	float finalval[MAXPOINTS + 2];
	float* finald;
	int size = (MAXPOINTS + 2) * sizeof(float),
		threadPerBlock = 1024,
		numBlock = tpoints/threadPerBlock + 1;
	hipMalloc(&finald,size);
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	//init_line();
	printf("Updating all points for all time steps...\n");
	//update();
	calc<<<numBlock,threadPerBlock>>>(finald,tpoints,nsteps);
	hipMemcpy(finalval, finald, size, hipMemcpyDeviceToHost);
	hipFree(finald);
	printf("Printing final results...\n");
	printfinal(finalval);
	printf("\nDone.\n\n");
	
	return 0;
}
